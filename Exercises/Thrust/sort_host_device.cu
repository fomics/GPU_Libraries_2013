#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <algorithm>
#include <cstdlib>

#include <hip/hip_runtime.h>


int main(int argc, char* argv[])
{
  size_t N = 10000; // Default value
  hipEvent_t start;
  hipEvent_t end;
  float elapsed_time;
    
  hipEventCreate(&start);
  hipEventCreate(&end);

  // generate 32M random numbers serially
  if (argc > 1) {
     N = atoi(argv[1]);
     std::cout << "Using number of elements = " << N << std::endl;
  }

  thrust::host_vector<int> h_vec(N);
  std::generate(h_vec.begin(), h_vec.end(), rand);
  thrust::device_vector<int> d_vec = h_vec;

  hipEventRecord(start,0);

  // starting sorting data on the host
  thrust::sort(h_vec.begin(), h_vec.end());
  // finished sorting data on the host

  hipEventSynchronize(end);
  hipEventRecord(end,0);
  hipEventSynchronize(end);
  hipEventElapsedTime(&elapsed_time, start, end);

  std::cout << "host sort took " << elapsed_time << " milliseconds" << std::endl;

  // output smallest/largest value
  std::cout << "Smallest value is\n" << h_vec[0] << std::endl;
  std::cout << "Largest value is\n" << h_vec[h_vec.size()-1] << std::endl;

  hipEventRecord(start,0);

  // starting sorting data on the host
  thrust::sort(d_vec.begin(), d_vec.end());
  // finished sorting data on the host

  hipEventSynchronize(end);
  hipEventRecord(end,0);
  hipEventSynchronize(end);
  hipEventElapsedTime(&elapsed_time, start, end);

  std::cout << "device sort took " << elapsed_time << " milliseconds" << std::endl;

  // output smallest/largest value
  std::cout << "Smallest value is\n" << d_vec[0] << std::endl;
  std::cout << "Largest value is\n" << d_vec[d_vec.size()-1] << std::endl;

  return 0;
}
